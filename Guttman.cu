#include "hip/hip_runtime.h"
__global__ void GuttmanPart1(double *D, double *projD, double *GT_B, int dataRows){
		unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
        int i = ix / (dataRows + 1), j = ix % (dataRows + 1);
        unsigned int idx;

        if (j > i) i = dataRows - i - 1, j = dataRows - j;
        idx = (i * dataRows + j);

		if (idx < (dataRows * (dataRows + 1) / 2)){	
			if (i != j){
				unsigned int idy = j * dataRows + i;
				//non-diagonal elements %
				if (projD[idx] != 0.0 ){
					GT_B[idx] = -D[idx] / projD[idx];
					GT_B[idy] = GT_B[idx];
				}else{
					GT_B[idx] = 0.0;
					GT_B[idy] = GT_B[idx];
				}
			}
		}
	}
}

__global__ void GuttmanPart2(double *GT_B, int dataRows){
		unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

		if (ix < dataRows){	
			// diagonal elements %
			GT_B[idx] = 0.0;

			for (k = 0; k < dataRows; k++){
				GT_B[idx] += GT_B[ix * dataRows + k];
			}
		}
	}
}