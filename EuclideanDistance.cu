
#include <hip/hip_runtime.h>
__global__ void EuclideanDistance(double *data, double *D, int dataRows, int dataCols){
	unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ix / (dataRows + 1), j = ix % (dataRows + 1);
	unsigned int idx;

	if (j > i) i = dataRows - i - 1, j = dataRows - j;
	idx = (i * dataRows + j);

	if (idx < (dataRows * (dataRows + 1) / 2))
	{
		double resta;
		double suma = 0;
		int k, m;

		for (k = 0 ; k < dataCols; k++) {
				resta = data[i * dataRows + k] - data[j * dataRows + k];
				suma += resta * resta;
		}
					
		suma = sqrt(suma);

		D[i * dataRows + j] = suma;
		D[j * dataRows + i] = suma;
	}
}